#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_cord_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    int x  = blockIdx.x * blockDim.x + threadIdx.x;
    int y  = blockIdx.y * blockDim.y + threadIdx.y;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord: (%3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index, x, y);
}

void print_cord(){
    int inputWidth = 4;

    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

int main() {
     print_cord();
    return 0;
}
