#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_cord_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    int x  = blockIdx.x * blockDim.x + threadIdx.x;
    int y  = blockIdx.y * blockDim.y + threadIdx.y;
    int z  = blockIdx.z * blockDim.z + threadIdx.z;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index, x, y, z);
}

void print_cord(){
    dim3 block(2, 2, 2);
    dim3 grid(2, 2, 2);

    print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

int main() {
     print_cord();
    return 0;
}
